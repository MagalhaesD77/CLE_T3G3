/**
 *  \file main.c (implementation file)
 *
 *  \brief Problem name: Bitonic sort.
 *  
 *
 *  \author Rafael Gil & Diogo Magalhães - May 2024
 */

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "common.h"
#include <hip/hip_runtime.h>

/**
 * \brief Read file and populate data array
 * 
 * \param fileName name of the file to read
 * \param lenNumberArray size of the array
 * \param numberArray array to be checked
*/
void readFile(char *fileName, int *lenNumberArray, int **numberArray);

/**
 * \brief checks if the number array is ordered in decreasing order
 * 
 * \param lenNumberArray size of the array
 * \param numberArray array to be checked
*/
void verifyIfSequenceIsOrdered(int lenNumberArray, int **numberArray);

/**
 *  \brief implementation of the imperitive bitonic sort - descending order.
 *
 *  \param array array of numbers to be sorted
 *  \param N length of the array
 *  \param startIndex index where sub-sequence starts
 *  \param endIndex index where sub-sequence ends
 */
_global static void bitonicSortOnGPU(int *array, int N, int startIndex, int endIndex);

/**
 *  \brief implementation of the imperitive bitonic sort - descending order.
 *
 *  \param array array of numbers to be sorted
 *  \param N length of the array
 *  \param startIndex index where sub-sequence starts
 *  \param endIndex index where sub-sequence ends
 */
void bitonicSortOnCPU(int *array, int N, int startIndex, int endIndex);

/**
 *  \brief Get the process time that has elapsed since last call of this time.
 *
 *  \return process elapsed time
 */

static double get_delta_time(void);

/**
 *  \brief Main thread.
 *
 *  Starts the simulation.
 *
 *  \param argc number of words of the command line
 *  \param argv list of words of the command line
 *
 *  \return status of operation
 */
int main(){
    // set up device
    int dev = 0;

    hipDeviceProp_t deviceProp;
    CHECK (hipGetDeviceProperties (&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK (hipSetDevice (dev)); // the gpu its going to use

    int n = 1024; // degree of a square matrix

    // host space
    int nElem = n * n; // number of elements of a square matrix
    int nBytes = nElem * sizeof(double); // matrix storage space in bytes
    double *sequence_gpu, *sequence_cpu;

    sequence_gpu = (double *)malloc(nBytes);
    sequence_cpu = (double *)malloc(nBytes);


    exit(EXIT_SUCCESS);
}

void readFile(char *fileName, int *lenNumberArray, int **numberArray) {
    FILE *file = fopen(fileName, "rb");
    if (file == NULL) {
        fprintf(stderr, "Unable to open the file.\n");
        exit(EXIT_FAILURE);
    }

    int err = fread(lenNumberArray, sizeof(int), 1, file);
    if (err <= 0) {
        fprintf(stderr, "Error while reading file.\n");
        exit(EXIT_FAILURE);
    }

    *numberArray = malloc(*lenNumberArray * sizeof(int));
    if (*numberArray == NULL) {
        fprintf(stderr, "Memory allocation failed.\n");
        exit(EXIT_FAILURE);
    }

    if (fread(*numberArray, sizeof(int), *lenNumberArray, file) != *lenNumberArray) {
        fprintf(stderr, "Error while reading numbers to array.\n");
        exit(EXIT_FAILURE);
    }

    fclose(file);
}

void verifyIfSequenceIsOrdered(int lenNumberArray, int **numberArray) {
    for (int i = 0; i < lenNumberArray - 1; i++) {
        if ((*numberArray)[i] < (*numberArray)[i + 1]) {
            printf("Error in position %d between elements %d and %d\n", i, (*numberArray)[i], (*numberArray)[i + 1]);
            return;
        }
    }

    printf("Everything is OK!\n");
}

void imperativeBitonicSort(int *array, int N, int startIndex, int endIndex){
    // iterate through the powers of 2 up to N
    // simulates the layers of the algorithm
    for (int k = 2; k <= N; k = 2 * k) {
        // iterate through half of the current value of k
        // controls the length of the comparison between the numbers
        for (int j = k / 2; j > 0; j = j / 2) {
            // iterates through the partition of the array
            for (int i = startIndex; i <= endIndex; i++) {
                int ij = i ^ j;     // bitwise XOR, to calculate the index where to perform the comparison
                if ((ij) > i) {     // assure correct order
                    if (((i & k) == 0                               // bitwise AND to check if i-th index is in the lower half of the bitonic sequence
                                && array[i] < array[ij])            // check if i-th element is smaller than ij
                        || ((i & k) != 0                            // bitwise AND to check if i-th index is in the upper half of the bitonic sequence
                                && array[i] > array[ij])) {         // check if i-th element is greater than ij

                        // performs a common swap between the elements of the array
                        int aux = array[i];
                        array[i] = array[ij];
                        array[ij] = aux;
                    }
                }
            }
        }
    }
}

static double get_delta_time(void){
  static struct timespec t0, t1;

  t0 = t1;

  if(clock_gettime (CLOCK_MONOTONIC, &t1) != 0){
    perror ("clock_gettime");
    exit(1);
  }

  return (double) (t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double) (t1.tv_nsec - t0.tv_nsec);
}